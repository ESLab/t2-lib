#include "hip/hip_runtime.h"

/*****************************************************************

    Copyright (C) 2013 Stefan Grönroos

    Authors: Stefan Grönroos <stefan.gronroos@abo.fi>

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Lesser General Public License for more details.

    You should have received a copy of the GNU Lesser General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.

********************************************************************/

#include "cuda_ldpc.h"


static cuda_ldpc_ctx h = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

/* CUDA PTX "assembler" instruction set contains a nice instruction for saturating conversion from one type to another,
   perfect for converting from short back to char */
__device__ char short_to_char(short value)
{
    int result;
    asm("cvt.sat.s8.s16 %0, %1;" : "=r" (result) : "h" (value));
    return (char)result;
}

void Check_CUDA_Error(const char *message)
{
    hipError_t error = hipGetLastError();

    if(error!=hipSuccess) {
        DEBUG("ERROR: %s: %s\n", message, hipGetErrorString(error) );
        exit(-1);
    }
}

__global__ void cuda_ldpc_init_messages(GPUMSG *emsg, const int num_edges) {
    int i = (blockIdx.x * blockDim.y + threadIdx.y);
    if(i < num_edges) {
        emsg[i].message[threadIdx.x].x = 0;
        emsg[i].message[threadIdx.x].y = 0;
        emsg[i].message[threadIdx.x].z = 0;
        emsg[i].message[threadIdx.x].w = 0;
    }
}


__global__ void cuda_ldpc_bn_update(GPUEDGE *hc, GPUEDGE *hb, DLLR* llr, int* col_map, int N, GPUMSG *emsg, int *col_idx) {
#define W threadIdx.x
#define MSG msg
    unsigned int i = (blockIdx.x * blockDim.y + threadIdx.y);
    int col_start;
    unsigned char counter = 0;
    short4 m;
    char4 msg;

    if(i < N)
    {
        col_start = col_idx[i];
        int index = col_start;
        i = (i<<5)+threadIdx.x;
        m.x = (short)llr[i].x;
        m.y = (short)llr[i].y;
        m.z = (short)llr[i].z;
        m.w = (short)llr[i].w;

        do
        {
            MSG = emsg[index].message[W];

#define BU1(X) \
    m.X = m.X + MSG.X;

            BU1(x);
            BU1(y);
            BU1(z);
            BU1(w);
#undef BU1
            index = hc[index].i_next;
            counter++;
        } while(index != col_start);

        counter = 0;
        do
        {
            MSG = emsg[index].message[W];

#define BU2(X) \
    MSG.X = short_to_char(m.X - MSG.X);

            BU2(x);
            BU2(y);
            BU2(z);
            BU2(w);
#undef BU2
            //Write back
            emsg[index].message[W] = MSG;

            counter++;
            index = hc[index].i_next;

        } while(index != col_start);

    }

}

__global__ void cuda_ldpc_bn_update_bitval(GPUEDGE *hc, GPUEDGE *hb, DBIT* bitval, DLLR* llr, int* col_map, int N, GPUMSG *emsg, int *col_idx) {
#define W threadIdx.x
#define MSG msg
    unsigned int i = (blockIdx.x * blockDim.y + threadIdx.y);
    int col_start;
    unsigned char counter = 0;
    short4 m;
    char4 msg;

    if(i < N)
    {
        col_start = col_idx[i];
        int index = col_start;
        i = (i<<5)+threadIdx.x;
        m.x = (short)llr[i].x;
        m.y = (short)llr[i].y;
        m.z = (short)llr[i].z;
        m.w = (short)llr[i].w;

        do
        {
            MSG = emsg[index].message[W];

#define BU1(X) \
    m.X += MSG.X;

            BU1(x);
            BU1(y);
            BU1(z);
            BU1(w);
#undef BU1

            index = hc[index].i_next;
            counter++;
        } while(index != col_start);

        counter = 0;
        do
        {
            MSG = emsg[index].message[W];
#define BU2(X) \
    MSG.X = short_to_char(m.X - MSG.X);

            BU2(x);
            BU2(y);
            BU2(z);
            BU2(w);
#undef BU2
            //Write back
            emsg[index].message[W] = MSG;

            counter++;
            index = hc[index].i_next;

        } while(index != col_start);

#define BITVAL_UPDATE(X) \
    bitval[i].X = (m.X >= 0 ? 0 : 1);

        BITVAL_UPDATE(x);
        BITVAL_UPDATE(y);
        BITVAL_UPDATE(z);
        BITVAL_UPDATE(w);

    }

}


__global__ void cuda_ldpc_cn_update(GPUEDGE *hc, GPUEDGE *hb, int M, GPUMSG *emsg, int *row_idx)
{

#define W threadIdx.x
#define MSG msg
    int i = (blockIdx.x * blockDim.y + threadIdx.y);

    int row_start;
    unsigned char counter = 0;
    unsigned char degree;
    uchar4 minMsg;
    char4 minLLR = {127, 127, 127, 127};
    char4 nMinLLR = {127, 127, 127, 127};
    char4 sign = {1,1,1,1};
    char4 msg;

    if (i < M) {
        row_start = row_idx[i]; //TODO: move to constant mem?
        int index = row_start;
        do {
            MSG = emsg[index].message[W];

#define CU1(X) \
    sign.X ^= MSG.X; \
    degree = fabsf(MSG.X); \
    nMinLLR.X = degree < minLLR.X ? minLLR.X : degree < nMinLLR.X ? degree : nMinLLR.X; \
    minLLR.X = degree < minLLR.X ? degree : minLLR.X; \
    minMsg.X = degree == minLLR.X ? counter : minMsg.X

            CU1(x);
            CU1(y);
            CU1(z);
            CU1(w);
#undef CU1
            index = hb[index].i_next;
            counter++;


        } while (index != row_start);

        counter = 0;

        do {
            MSG = emsg[index].message[W];
#define CU2(X) \
    MSG.X = (1-(((sign.X^MSG.X) & 0x80) >> 6)) * (counter != minMsg.X ? minLLR.X : nMinLLR.X)

            CU2(x);
            CU2(y);
            CU2(z);
            CU2(w);
#undef CU2

            //Write back
            emsg[index].message[W] = MSG;
            index = hb[index].i_next;
            counter++;
        } while(index != row_start);
    }
}


__global__ void cuda_ldpc_check_satisfied(GPUEDGE *hb, DBIT *bitval, int *llr_map, int *unsatisfied, int M, int *row_idx) {
    int i = (blockIdx.x * blockDim.y + threadIdx.y);
    if (i == 0) unsatisfied[0] = 0;
    __syncthreads();
    if (i < M) {
        int row_start = row_idx[i];
        int index = row_start;
        short sum = 0;
        do {
            int tmp = llr_map[index]*CODEWORDS+threadIdx.x;
            sum ^= bitval[tmp].x;
            sum ^= bitval[tmp].y;
            sum ^= bitval[tmp].z;
            sum ^= bitval[tmp].w;

            index = hb[index].i_next;
        } while (index != row_start);
        if (sum==1) atomicAdd(unsatisfied, 1);
    }
}

void cuda_ldpc_load(GPUEDGE *hb_host, GPUEDGE *hc_host, int numEdges, int M, int N, int* llr_map_h, int* col_map_h, int* row_idx_h, int* col_idx_h) {
    CUDA_SAFE_CALL(hipMalloc((void**)&h.hc_dev, numEdges*sizeof(GPUEDGE)));
    CUDA_SAFE_CALL(hipMalloc((void**)&h.hb_dev, numEdges*sizeof(GPUEDGE)));
    CUDA_SAFE_CALL(hipMalloc((void**)&h.llr_map_d, numEdges*sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&h.edge_msg, numEdges*sizeof(GPUMSG)));

    CUDA_SAFE_CALL(hipMalloc((void**)&h.row_idx_d, M*sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&h.col_idx_d, N*sizeof(int)));

    CUDA_SAFE_CALL(hipMalloc((void**)&h.llr_d, CODEWORDS*N*sizeof(DLLR))); //might as well allocate space for llrs already
    CUDA_SAFE_CALL(hipMalloc((void**)&h.bitval_d, CODEWORDS*N*sizeof(DBIT)));
    CUDA_SAFE_CALL(hipMalloc((void**)&h.unsatisfied_d, 1*sizeof(int)));

    CUDA_SAFE_CALL(hipMemcpy(h.hb_dev, hb_host, numEdges*sizeof(GPUEDGE), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(h.hc_dev, hc_host, numEdges*sizeof(GPUEDGE), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(h.llr_map_d, llr_map_h, numEdges*sizeof(int), hipMemcpyHostToDevice));

    CUDA_SAFE_CALL(hipMemcpy(h.row_idx_d, row_idx_h, M*sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(h.col_idx_d, col_idx_h, N*sizeof(int), hipMemcpyHostToDevice));

    h.cuda_num_edges = numEdges;
    if (h.llr_d)
        DEBUG("Cuda Load done\n");
}


/* A decoder function using the kernels above.
  Input LLRs are in llr_h_words, and are assumed to be ordered such that
  llr_h_words[b*PARALLEL_FEC_BLOCKS/4 + c] contains bit b of codeword c.

  Returns the result to the (assumedly already allocated) bitval_h_words
  in the same memory layout for further reordering in calling function, if necessary
*/

void cuda_ldpc_d(DLLR *llr_h_words, int M, int N, DBIT *bitval_h_words, int max_iter) {
    int unsatisfied = -1;
    if (max_iter < 1) max_iter = 1;
    START_CLOCK(2)
            CUDA_SAFE_CALL(hipMemcpy(h.llr_d, llr_h_words, CODEWORDS*N*sizeof(DLLR), hipMemcpyHostToDevice));

    dim3 blockSize(CODEWORDS,EDGESPERBLOCK);
    dim3 blockSizeCN(CODEWORDS,EDGESPERBLOCK);
    int gridSizeBN = ceil((float)N / EDGESPERBLOCK);
    int gridSizeCN = ceil((float)M / EDGESPERBLOCK);
    int gridSizeInit = ceil((float)h.cuda_num_edges / EDGESPERBLOCK);

    cuda_ldpc_init_messages<<<gridSizeInit, blockSize>>>(h.edge_msg, h.cuda_num_edges);

    /* We hardly use any shared memory, so maximize L1 cache size instead */
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuda_ldpc_bn_update), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuda_ldpc_cn_update), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuda_ldpc_check_satisfied), hipFuncCachePreferL1);


    for (int i=0; i < max_iter; i++) {
        Check_CUDA_Error("Before BN update");

        if(i % 10 != 0 || i == 0) {
            cuda_ldpc_bn_update<<<gridSizeBN, blockSize>>>(h.hc_dev, h.hb_dev, h.llr_d, h.col_map_d, N, h.edge_msg, h.col_idx_d);
        } else {
            cuda_ldpc_bn_update_bitval<<<gridSizeBN, blockSize>>>(h.hc_dev, h.hb_dev, h.bitval_d, h.llr_d, h.col_map_d, N, h.edge_msg, h.col_idx_d);
            cuda_ldpc_check_satisfied<<<gridSizeCN, blockSize>>>(h.hb_dev, h.bitval_d, h.llr_map_d, h.unsatisfied_d, M, h.row_idx_d);
            CUDA_SAFE_CALL(hipMemcpy(&unsatisfied, h.unsatisfied_d, 1*sizeof(int), hipMemcpyDeviceToHost));
            if (unsatisfied == 0) {DEBUG("CUDA done already after %i iterations!\n", i); break;}
        }

        cuda_ldpc_cn_update<<<gridSizeCN, blockSizeCN>>>(h.hc_dev, h.hb_dev, M, h.edge_msg, h.row_idx_d);
    }
    cuda_ldpc_bn_update_bitval<<<gridSizeBN, blockSize>>>(h.hc_dev, h.hb_dev, h.bitval_d, h.llr_d, h.col_map_d, N, h.edge_msg, h.col_idx_d);

    CUDA_SAFE_CALL(hipMemcpy(bitval_h_words, h.bitval_d, CODEWORDS*N*sizeof(DBIT), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    BENCHMARK_NOW(2, "CUDA ldpc_d")

}


/* Clear device memory */

void cuda_ldpc_destroy() {
    DEBUG("Freeing CUDA memory\n");
    if (h.hc_dev) CUDA_SAFE_CALL(hipFree(h.hc_dev));
    if (h.hb_dev) CUDA_SAFE_CALL(hipFree(h.hb_dev));
    if (h.llr_d) CUDA_SAFE_CALL(hipFree(h.llr_d));
    if (h.bitval_d) CUDA_SAFE_CALL(hipFree(h.bitval_d));
    if (h.llr_map_d) CUDA_SAFE_CALL(hipFree(h.llr_map_d));
    if (h.unsatisfied_d) CUDA_SAFE_CALL(hipFree(h.unsatisfied_d));
    if (h.edge_msg) CUDA_SAFE_CALL(hipFree(h.edge_msg));

    if (h.row_idx_d) CUDA_SAFE_CALL(hipFree(h.row_idx_d));
    if (h.col_idx_d) CUDA_SAFE_CALL(hipFree(h.col_idx_d));

    memset(&h, 0, sizeof(cuda_ldpc_ctx)); /* NULL the context */
}
